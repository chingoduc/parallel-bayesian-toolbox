
#include <hip/hip_runtime.h>
/**
  * GPU based mean calculation
  */
extern "C" __global__ void pbtCalcMeanCUDA(float* data, float* weights, int stateDimension, int numberOfSamples, float* estimation){
    int dimension = blockIdx.x;
    if (dimension < stateDimension){
        estimation[dimension] = 0;
        for (unsigned int i = 0; i < numberOfSamples; ++i){
            estimation[dimension] += data[dimension + (i*stateDimension)] * weights[dimension];
        }
    }
}


void callCalcMeanKernel(float* data, float* weights, int stateDimension, int numberOfSamples, float* estimation){
    //call kernel
    pbtCalcMeanCUDA<<<stateDimension,1>>>(data,weights,stateDimension,numberOfSamples,estimation);
}
